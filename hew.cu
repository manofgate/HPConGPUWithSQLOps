#include <stdio.h>
#include <hip/hip_runtime.h>
#include <string>
//using namespace std;
__global__ void myKernel(int* c,int N ) 
{ if(threadIdx.x <N){
  c[threadIdx.x] = 2;
	printf("Hello, world from the device! \n"); 
//__syncthreads();
}
} 

int main() 
{ 
//int dayName[] = {1, 1,1,1,1,1,1,1,2,2};
int* dayName = (int*)malloc(10*sizeof(int));
for(int i=0; i<10; i++){
//printf("dayName spot %d , %d \n", i, dayName[i]);
 dayName[i]= 1;
}
int* d_c;
hipMalloc((void **)&d_c,10*sizeof(int));
hipMemcpy(d_c, dayName, 10*sizeof(int), hipMemcpyHostToDevice);
//printf("first is %s \n", d_c[1]);

//cudaPrintfInit();   
myKernel<<<1,10>>>(d_c, 10); 
//cudaPrintfDisplay(stdout, true); 
//cudaPrintfEnd();
hipDeviceSynchronize();
hipMemcpy(dayName, &d_c, 10*sizeof(int), hipMemcpyDeviceToHost );
hipFree(d_c);
for(int i=0; i<10; i++){
printf("dayName spot %d , %d  \n", i, dayName[i]);
}
printf("I am saine \n");
} 
